#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/binary_search.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
  
#if 0 // potential optimization
#include <thrust/experimental/cuda/pinned_allocator.h> 
typedef thrust::experimental::cuda::pinned_allocator<double> AllocPinned;
thrust::device_vector< double, AllocPinned> pinned_host_vec(num);
T *h_ptr = pinned_host_vec.data();
T *raw_d_ptr = 0;
hipHostGetDevicePointer(&raw_d_ptr, h_ptr, 0);
thrust::device_ptr<T> dt_ptr(raw_dt_ptr);
thrust::host_ptr<T> ht_ptr(h_ptr,h_ptr+num);
#endif


#define DEBUG_CPU 0
#define SQR(x) ((x)*(x))

// A bunch of junk to deal with querying GPU info
namespace ebf_cuda {
/**
        \brief Unrecoverable error exception.

        Throw an instance of this class to indicate an unrecoverable error
        was encountered. Do not throw it directly, but through the use of ERROR() macro.
*/
class runtime_error : public std::runtime_error
{
public:
        runtime_error(const std::string &msg) : std::runtime_error(msg) {}
        virtual ~runtime_error() throw() {};
};

#ifndef THROW_IS_ABORT
        #define ERROR(msg) throw ebf_cuda::runtime_error(msg);
#else
        #define ERROR(msg) { fprintf(stderr, "%s\n", std::string(msg).c_str()); abort(); }
#endif


/*!  Unrecoverable CUDA error, thrown by cudaErrCheck macro.
 *    Do not use directly. use cudaErrCheck macro instead.
 */
struct cudaException : public ebf_cuda::runtime_error
{
        cudaException(hipError_t err) : ebf_cuda::runtime_error( hipGetErrorString(err) ) {}

        static void check(hipError_t err, const char *fun, const char *file, const int line) {
                if(err != hipSuccess)
                        throw cudaException(err);
        }
};
/**
 *      cudaErrCheck macro -- aborts with message if the enclosed call returns != hipSuccess
 */
#define cudaErrCheck(expr) \
        cudaException::check(expr, __PRETTY_FUNCTION__, __FILE__, __LINE__)

// selects GPU to use and returns gpu ID or -1 if using CPU
int init_cuda() 
{ 
    // Select the proper device
    const char* devstr = getenv("CUDA_DEVICE");
    const int env_dev = (devstr != NULL) ? atoi(devstr) : 0;
    int dev = env_dev;
    int devcnt; ebf_cuda::cudaErrCheck( hipGetDeviceCount(&devcnt) );
    if( dev >= 0 && dev < devcnt )
       { 
       ebf_cuda::cudaErrCheck( hipSetDevice(dev) ); 
       hipDeviceSetCacheConfig(hipFuncCachePreferL1);
       }
    else
       {
        dev = -1;
       	std::cerr << "# Cannot select the CUDA device. Using CPU!" << std::endl;
	}
    return dev;
}

} // namespace ebf_cuda


// global variables for timing code
uint memoryTime, kernelComputeTime, kernelSortTime;
int verbose;

namespace limbdarkening {

struct ellke_functor
{
   __host__ __device__ inline thrust::tuple<double,double> operator()(const double k)
      {
      double m1 = 1.0-k*k;
      double logm1 = log(m1);
      double ek, kk;
	{
	const double a1=0.44325141463;
	const double a2=0.06260601220;
	const double a3=0.04757383546;
	const double a4=0.01736506451;
	const double b1=0.24998368310;
	const double b2=0.09200180037;
	const double b3=0.04069697526;
	const double b4=0.00526449639;
	const double ee1=1.+m1*(a1+m1*(a2+m1*(a3+m1*a4)));
	const double ee2=m1*(b1+m1*(b2+m1*(b3+m1*b4)))*(-logm1);
	ek = ee1+ee2;
	}	
	{
	const double a0=1.38629436112;
	const double a1=0.09666344259;
	const double a2=0.03590092383;
	const double a3=0.03742563713;
	const double a4=0.01451196212;
	const double b0=0.5;
	const double b1=0.12498593597;
	const double b2=0.06880248576;
	const double b3=0.03328355346;
	const double b4=0.00441787012;
	const double ek1=a0+m1*(a1+m1*(a2+m1*(a3+m1*a4)));
	const double ek2=(b0+m1*(b1+m1*(b2+m1*(b3+m1*b4))))*logm1;
	kk = ek1-ek2;
	}
      return thrust::make_tuple(ek,kk);
      }
};

__host__ __device__ inline double ellpic_bulirsch(const double n, const double k)
{
   double kc = sqrt(1.-k*k);
   double p = n+1.0;
#if CPU_DEBUG
    assert(p>=0.);
#endif
    double m0 = 1.0;
    double c =1.0;
    double d = rsqrt(p);
    p = 1.0/d;
    double e = kc;
    do {
       double f = c;
       c = d/p+c;
       double g = e/p;
       d = 2.*(f*g+d);
       p = g + p;
       g = m0;
       m0 = kc + m0;
       if(SQR(1.0-kc/g)>1.e-16)
       { kc = 2.*sqrt(e);   e = kc*m0; }
       else
       { return 0.5*M_PI*(c*m0+d)/(m0*(m0+p)); }
    } while(true);
}

#define ACCEL_TRIVIAL_RETURN 1
#define GROUP_FUNC_CALLS_MINI 1
#define GROUP_FUNC_CALLS 0 // doesn't help performance

template<bool Uniform>
struct occultquad_functor
{
	 static const double tol = 1.e-14;
	 static const int  z_index = 0;
	 static const int u1_index = 1;
	 static const int u2_index = 2;
	 static const int p0_index = 3;
	 static const int muo1_index = 4;
	 static const int mu0_index = 5;

	 // could optimize further, by multiple if statements, so that only call ellke and ellpic_bulirsch in one place
	 // 
	__host__ __device__ inline 
	// thrust::tuple<double,double> 
	void
	operator()( thrust::tuple< const double&, const double&, const double&, const double&, double&, double& >  val)
	 {
	 ellke_functor ellke;
	 double z  = thrust::get<z_index>(val);
	 const double p0 = thrust::get<p0_index>(val);
	 const double p = fabs(p0); // "to mesh with fitting routines"
#if ACCEL_TRIVIAL_RETURN 
	 if((p<=0.) || (z>=1.+p)) // case 0, 1
	   {
	   thrust::get<muo1_index>(val) = 1.;
	   if(Uniform) thrust::get<mu0_index>(val) = 1.;
	   return ;
	   }
#endif
	 const double u1 = thrust::get<u1_index>(val);
	 const double u2 = thrust::get<u2_index>(val);
	 const double omega = 1.0-(u1-0.5*u2)/3.;
	 double lambdad = 0.;
	 double lambdae = 0.;
	 double etad = 0.;

	 z = (fabs(p-z)<tol) ? p : z;
	 z = (fabs((p-1.)-z)<tol) ? p-1. : z;
	 z = (fabs((1.-p)-z)<tol) ? 1.-p : z;
	 z = (z<tol) ? 0. : z;
	 const double x1 = (p-z)*(p-z);
	 const double x2 = (p+z)*(p+z);

#if !ACCEL_TRIVIAL_RETURN 
         // case 0 and 1 moved up to reduce memory loads
	 if(p<=0.)  // case 0
	   {
	   thrust::get<muo1_index>(val) = 1.;
	   if(Uniform) thrust::get<mu0_index>(val) = 1.;
	   return ;
	   }
	 else if(z>=1.+p) // case 1 // source is unocculted (why so much code?)
	   {
	   thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	   if(Uniform) thrust::get<mu0_index>(val)  = 1.-lambdae;
	   return ;
	   }
	 else 
#endif
	 if( (p>=1.) && (z<=p-1.) ) // case 11 (source completely occulted)
	   {
	   etad = 0.5;
	   lambdae = 1.;
	   thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	   if(Uniform) thrust::get<mu0_index>(val)  = 1.-lambdae;
//	   return ;
	   }
	else // partially occulted
	{
	if( (z>=fabs(1.-p)) && (z<1.+p) ) // case 2,7,8 (during ingress/egress)
	   {
	   double tmp1 = (1.-p*p+z*z)/(2.*z);
	   if(tmp1>1.) tmp1 = 1.;	   if(tmp1<-1.) tmp1 = -1.;
	   double kap1 = acos(tmp1);
	   double tmp2 = (p*p+z*z-1.)/(2.*p*z);
	   if(tmp2>1.) tmp2 = 1.;	   if(tmp2<-1.) tmp2 = -1.;
	   double kap0 = acos(tmp2);
	   double tmp3 = 4.*z*z-SQR(1.+z*z-p*p);
//	   double tmp3 = 4.*z*z-(1.+z*z-p*p)*(1.+z*z-p*p);
	   if(tmp3<0.) tmp3 = 0.;
	   lambdae = (p*p*kap0+kap1-0.5*sqrt(tmp3))/M_PI;
	   etad = (kap1+p*p*(p*p+2.*z*z)*kap0-0.25*(1.+5.*p*p+z*z)*sqrt((1.-x1)*(x2-1.)))/(2.*M_PI);
	   // don't return here!
	   }


#if GROUP_FUNC_CALLS 
         // I thought it would be good to parallelize computation of Ek, Kk and elliptic integral
	 // But on test case, it's slower, so I abandonded the idea
	 double q, n;
	 bool compute_EkKk = false, compute_bs = false;
	 if(z==p) // case 5, 6, 7 (edge of planet at origin of star)
	   {  q = (p<=0.5) ? 2.*p : 0.5/p;  compute_EkKk = true; }
	 else if( ((z>0.5+fabs(p-0.5)) && (z<1.+p)) || ((p>0.5) && (z>fabs(1.-p)) && (z<p) ) ) // case 2, 8 (during ingress/egress) (needs etad from uniform disk code)
	   {  
	   q = sqrt((1.-x1)/(4.*p*z));
	   n = 1./x1-1.;          
   	   compute_EkKk = true; compute_bs = true; 
	   }
	 else if((p<1.)&&(z!=1.-p)&&(z!=0.)) // case 3, 9 (planet completely inside star)
	   { 
	   q = rsqrt((1.-x1)/(x2-x1)); 
	   n = x2/x1-1.;
   	   compute_EkKk = true; compute_bs = true; 
	   }
	 thrust::tuple<double,double> EkKk = (compute_EkKk) ? ellke(q) : thrust::make_tuple(0.,0.);
	 double ellpic_bulrisch_n_q = (compute_bs) ? ellpic_bulirsch(n,q) : 0.;
#endif	 
 
	 if(z==p) // case 5, 6, 7 (edge of planet at origin of star)
	   {
#if GROUP_FUNC_CALLS_MINI
	   double q = (p<=0.5) ? 2.*p : 0.5/p;
	   thrust::tuple<double,double> EkKk = (p!=0.5) ? ellke(q) : thrust::make_tuple(0.,0.);
#endif
	   if(p<0.5) // case 5
	     {
#if !GROUP_FUNC_CALLS && !GROUP_FUNC_CALLS_MINI
	     double q = 2.*p;
	     thrust::tuple<double,double> EkKk = ellke(q);
#endif
	     lambdad = 1./3.+2.*(4.*(2.*p*p-1.)*EkKk.get<0>()+(1.-4.*p*p)*EkKk.get<1>())/(9.*M_PI);
	     etad = 0.5*p*p*(p*p+2.*z*z);
	     lambdae = p*p;
	     }
	   else if( p>0.5) // case 7 (need etad from uniform disk code)
	     {
#if !GROUP_FUNC_CALLS && !GROUP_FUNC_CALLS_MINI
	     double q = 0.5/p;
	     thrust::tuple<double,double> EkKk = ellke(q);
#endif
	     lambdad = 1./3.+(16.*p*(2.*p*p-1.)*EkKk.get<0>()-
	       (32.*p*p*p*p-20.*p*p+3.)/(p)*EkKk.get<1>())/(9.*M_PI);
	     }
	   else // case 6
	     {
	     lambdad = 1./3.-4./(9.*M_PI);
	     etad = 3./32.;
	     }
	   thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	   if(Uniform) thrust::get<mu0_index>(val)  = 1.-lambdae;
//	   return; 
	   }
	 else if( ((z>0.5+fabs(p-0.5)) && (z<1.+p)) || ((p>0.5) && (z>fabs(1.-p)) && (z<p) ) ) // case 2, 8 (during ingress/egress) (needs etad from uniform disk code)
	   {
#if !GROUP_FUNC_CALLS
	   const double q = sqrt((1.-x1)/(4.*p*z));
	   thrust::tuple<double,double> EkKk = ellke(q);
	   const double n = 1./x1-1.;          
//	   const double n = 1./(p-z);  // from python version?!?
	   const double ellpic_bulrisch_n_q = ellpic_bulirsch(n,q);
#endif
	   const double x3 = p*p-z*z;
	   lambdad = 1./(9.*M_PI)*rsqrt(p*z)*
	     ( ((1.-x2)*(2.*x2+x1-3.)-3.*x3*(x2-2.))*EkKk.get<1>()
	     +4.*p*z*(z*z+7.*p*p-4.)*EkKk.get<0>()
	     -3.*x3/x1*ellpic_bulrisch_n_q );
	   thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	   if(Uniform) thrust::get<mu0_index>(val)  =1.- lambdae;
//	   return; 
	   }
	 else if(p<1.) // case 3, 4, 9, 10 (planet completely inside star)
	   {
#if CPU_DEBUG
	   assert(z<1.-p); 
#endif
	     etad = 0.5*p*p*(p*p+2.*z*z);
	     lambdae = p*p;
	     if(z==1.-p) // case 4
	       {
	       lambdad = ( 6.*acos(1.-2.*p)-4.*sqrt(p*(1.-p))*(3.+2.*p-8.*p*p) )/(9.*M_PI);
	       if(p>0.5)
	        lambdad -= 2./3.;
	       thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	       if(Uniform) thrust::get<mu0_index>(val)  = 1.-lambdae;
//	       return;
	       }
	     else if(z==0.) // case 10
	       {
	       lambdad = -2./3.*(1.-p*p)*sqrt(1.-p*p);
	       thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	       if(Uniform) thrust::get<mu0_index>(val)  = 1.-lambdae;
//	       return;
	       }
	     else  // case 3, 9
	       {
#if !GROUP_FUNC_CALLS
	       double q = rsqrt((1.-x1)/(x2-x1));
	       thrust::tuple<double,double> EkKk = ellke(q);	   
	       double n = x2/x1-1.;
	       double ellpic_bulrisch_n_q = ellpic_bulirsch(n,q);
#endif
	       double x3 = p*p-z*z;
	       lambdad = 2./(9.*M_PI)*rsqrt(1.-x1)*
                       ( (1.-5.*z*z+p*p+x3*x3)*EkKk.get<1>()
		         +(1.-x1)*(z*z+7.*p*p-4.)*EkKk.get<0>()
			 -3.*x3/x1*ellpic_bulirsch(n,q) );
	       thrust::get<muo1_index>(val) = 1.-((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
	       if(Uniform) thrust::get<mu0_index>(val)  = 1.-lambdae;
//	       return;			    
	       }
	     }  // end if (p<1.) case 3,4,9,10
	   } // end else case partially occulted

	double tmp = ((1.-u1-2.*u2)*lambdae+(u1+2.*u2)*(lambdad+2./3.*(p > z))+u2*etad)/omega;
        thrust::get<muo1_index>(val) = 1.+ (1.-2.*(p0>0.)) * tmp;
        if(Uniform) thrust::get<mu0_index>(val) = 1.+ (1.-2.*(p0>0.)) * lambdae;
        return;

	}
};


	template<bool Uniform>
	void occultquad_wrapper_c(const double *ph_z, const double *ph_u1, const double *ph_u2, const double *ph_p, const int num, double *ph_muo1, double *ph_mu1)
	  {

	    int gpuid = ebf_cuda::init_cuda();
	    // put vectors in thrust format from raw points
	    thrust::host_vector<double> h_z(ph_z,ph_z+num);
	    thrust::host_vector<double> h_u1(ph_u1,ph_u1+num);
	    thrust::host_vector<double> h_u2(ph_u2,ph_u2+num);
	    thrust::host_vector<double> h_p(ph_p,ph_p+num);

	    cutCreateTimer(&memoryTime);  cutCreateTimer(&kernelComputeTime);	cutCreateTimer(&kernelSortTime);
	    cutResetTimer(memoryTime);    cutResetTimer(kernelComputeTime);	cutResetTimer(kernelSortTime);


	    if(gpuid>=0)
	    	{
		// allocate mem on GPU
		thrust::device_vector<double> d_z(num);
		thrust::device_vector<double> d_u1(num);
		thrust::device_vector<double> d_u2(num);
		thrust::device_vector<double> d_p(num);
		thrust::device_vector<double> d_muo1(num);
		thrust::device_vector<double> d_mu1(num);
		hipDeviceSynchronize();

		cutStartTimer(memoryTime);
		// transfer input params to GPU
		d_z = h_z;
		d_u1 = h_u1;
		d_u2 = h_u2;
		d_p = h_p;
		hipDeviceSynchronize();
		cutStopTimer(memoryTime);

		// distribute the computation to the GPU
		cutStartTimer(kernelComputeTime);

		thrust::for_each(
	   	   thrust::make_zip_iterator(thrust::make_tuple(d_z.begin(),d_u1.begin(),d_u2.begin(),d_p.begin(), d_muo1.begin(),d_mu1.begin() )),
	   	   thrust::make_zip_iterator(thrust::make_tuple(d_z.end(),  d_u1.end(),  d_u2.end(),  d_p.end(),   d_muo1.end(),  d_mu1.end()   )), 
		   occultquad_functor<Uniform>() );

		 hipDeviceSynchronize();
		 cutStopTimer(kernelComputeTime);

		 // transfer results back to host
		 cutStartTimer(memoryTime);
		 thrust::copy(d_muo1.begin(),d_muo1.end(),ph_muo1);
		 if(Uniform) thrust::copy(d_mu1.begin(), d_mu1.end(), ph_mu1);
		 hipDeviceSynchronize();
		 cutStopTimer(memoryTime);
		 }
	       else
		 {
		 // distribute the computation to the CPU
		 cutStartTimer(kernelComputeTime);

		 thrust::for_each(
		    	   	   thrust::make_zip_iterator(thrust::make_tuple(h_z.begin(),h_u1.begin(),h_u2.begin(),h_p.begin(),ph_muo1,ph_mu1)),
				   thrust::make_zip_iterator(thrust::make_tuple(h_z.end(),  h_u1.end(),  h_u2.end(),  h_p.end(),ph_muo1+num,ph_mu1+num)), 
				   occultquad_functor<Uniform>() );
		  cutStopTimer(kernelComputeTime);	
		  }
	}

} // end namespace limbdarkening


int main(int argc, char **argv)
{
	int num_zs = 8196;
	int num_planet_sizes = 1024;
	double max_planet_size = 0.1;
	verbose = 0;
	   { // read parameters from command line
	   std::istringstream iss;
	   if(argc>1)
	     {
	     iss.str(std::string (argv[1]));
	     iss >> num_zs;
	     iss.clear();
	     }
	   if(argc>2)
	     {
	     iss.str(std::string (argv[2]));
	     iss >> num_planet_sizes;
	     iss.clear();
	     }
	   if(argc>3)
	     {
	     iss.str(std::string (argv[3]));
	     iss >> max_planet_size;
	     iss.clear();
	     }
	   if(argc>4)
	     {
	     iss.str(std::string (argv[4]));
	     iss >> verbose;
	     iss.clear();
	     }
	   }
	int num_eval = num_planet_sizes*num_zs;  
	std::cerr << "# " << argv[0] << " nzs= " << num_zs << " nps= " << num_planet_sizes << " max_planet_size= " << max_planet_size << " verbose= " << verbose << "\n";

	// allocate host memory
	thrust::host_vector<double> h_z(num_eval);
	thrust::host_vector<double> h_u1(num_eval);
	thrust::host_vector<double> h_u2(num_eval);
	thrust::host_vector<double> h_p(num_eval);
	thrust::host_vector<double> h_muo1(num_eval);
	thrust::host_vector<double> h_mu0(num_eval);

	// initialize data on host 
	for(int i=0;i<num_planet_sizes;++i)
	  {
	  for(int j=0;j<num_zs;++j)
	    {	
	    int k = i*num_zs+j;
	    if(verbose>=256)
	       h_z[k] = 2.0*rand()/RAND_MAX;
	    else
	       h_z[k] = 2.0*static_cast<double>(j)/static_cast<double>(num_zs);


	    h_u1[k] = 0.1;
	    h_u2[k] = 0.3;
	    h_p[k] = max_planet_size*static_cast<double>(i+1)/static_cast<double>(num_planet_sizes);
	    }
	  }

	// optional check up on input values
	if(verbose%128>4) 
	   {
	   for(int i = 0; i < num_eval; i++)
              std::cout << " i= " << i << "z= " << h_z[i] << " u1= " << h_u1[i] << " u2= " << h_u2[i] << " p= " << h_p[i] << std::endl;
	   }

	// extract raw pointers to host memory to simulate what you'd get from IDL or another library
	double *ph_z = &h_z[0];
	double *ph_u1 = &h_u1[0];
	double *ph_u2 = &h_u2[0];
	double *ph_p  = &h_p[0];
	double *ph_muo1 = &h_muo1[0];
	double *ph_mu0 = &h_mu0[0];

	// wrapper function that could be called from IDL
	if(verbose%256>=128) // testing whether merging function calls helps
	   limbdarkening::occultquad_wrapper_c<true>(ph_z,ph_u1,ph_u2,ph_p,num_eval,ph_muo1,ph_mu0);
	else
	   limbdarkening::occultquad_wrapper_c<false>(ph_z,ph_u1,ph_u2,ph_p,num_eval,ph_muo1,ph_mu0);

	 // print results
	 if(verbose%128>0 ) 
	    {
	    for(int i = 0; i < num_eval; i++)
		{
		 std::cout << "i= " << i << " z= " << h_z[i] << " p= " << h_p[i] << " muo1= " << ph_muo1[i];
		 if(verbose>=128)
		    std::cout << " mu1= " << ph_mu0[i];
		 std::cout << std::endl;
		}
            }
	
	// report time spent on calculations and memory transfer
	std::cerr << "# Time for compute kernel: " << cutGetTimerValue(kernelComputeTime) << " ms, Time for sort kernels: " << cutGetTimerValue(kernelSortTime) << " ms, Time for memory: " << cutGetTimerValue(memoryTime) << " ms, Total time: " << cutGetTimerValue(kernelComputeTime)+cutGetTimerValue(kernelSortTime)+cutGetTimerValue(memoryTime) << " ms \n"; 
	return 0;
}



