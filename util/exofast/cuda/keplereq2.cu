#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <string>
#include <sstream>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/for_each.h>
  
#define DEBUG_CPU 0

// A bunch of junk to deal with querying GPU info
namespace ebf {
/**
        \brief Unrecoverable error exception.

        Throw an instance of this class to indicate an unrecoverable error
        was encountered. Do not throw it directly, but through the use of ERROR() macro.
*/
class runtime_error : public std::runtime_error
{
public:
        runtime_error(const std::string &msg) : std::runtime_error(msg) {}
        virtual ~runtime_error() throw() {};
};

#ifndef THROW_IS_ABORT
        #define ERROR(msg) throw ebf::runtime_error(msg);
#else
        #define ERROR(msg) { fprintf(stderr, "%s\n", std::string(msg).c_str()); abort(); }
#endif


/*!  Unrecoverable CUDA error, thrown by cudaErrCheck macro.
 *    Do not use directly. use cudaErrCheck macro instead.
 */
struct cudaException : public ebf::runtime_error
{
        cudaException(hipError_t err) : ebf::runtime_error( hipGetErrorString(err) ) {}

        static void check(hipError_t err, const char *fun, const char *file, const int line) {
                if(err != hipSuccess)
                        throw cudaException(err);
        }
};
/**
 *      cudaErrCheck macro -- aborts with message if the enclosed call returns != hipSuccess
 */
#define cudaErrCheck(expr) \
        cudaException::check(expr, __PRETTY_FUNCTION__, __FILE__, __LINE__)

}

// global variables for timing code
uint memoryTime, kernelTime;


// Code that gets turned into a GPU kernel by thrust
struct keplereq_functor
{
	static const double del_sq = 1.0e-12;
    	static const double k = 0.85;
	static const int num_max_it = 20;
	static const double third = 1.0/3.0;

	keplereq_functor() { };

#if !DEBUG_CPU
	__device__ 
#endif
	__host__ inline void operator()( thrust::tuple<const double&,const double&, double&>  val )
	{
	double M = thrust::get<0>(val);
	double e = thrust::get<1>(val);
#if DEBUG_CPU
	assert(M>=0.);
	assert(M<=2.*M_PI);
	assert(e>=0.);
	assert(e<=1.);
#endif
 	double x = (M<M_PI) ? M + k*e : M - k*e;
    	double F = 1.;
    	for(int i=0;i<num_max_it;++i)
	   {
	   double es, ec;
 	   sincos(x,&es,&ec);
           es *= e;
           F = (x-es)-M;
           if(fabs(F)<del_sq) break;
           ec *= e;
 	   const double Fp = 1.-ec;
           const double Fpp = es;
           const double Fppp = ec;
           double Dx = -F/Fp;
           Dx = -F/(Fp+0.5*Dx*Fpp);
       	   Dx = -F/(Fp+0.5*Dx*(Fpp+third*Dx*Fppp));
           x += Dx;
      	   }
	thrust::get<2>(val) = x;
	};
};


// selects GPU to use and returns gpu ID or -1 if using CPU
int init_cuda() 
{ 
    // Select the proper device
    const char* devstr = getenv("CUDA_DEVICE");
    const int env_dev = (devstr != NULL) ? atoi(devstr) : 0;
    int dev = env_dev;
    int devcnt; ebf::cudaErrCheck( hipGetDeviceCount(&devcnt) );
    if( dev >= 0 && dev < devcnt )
       { 
       ebf::cudaErrCheck( hipSetDevice(dev) ); 
       hipDeviceSetCacheConfig(hipFuncCachePreferL1);
       }
    else
       {
        dev = -1;
       	std::cerr << "# Cannot select the CUDA device. Using CPU!" << std::endl;
	}
    return dev;
}


// keplereq_wrapper_C:
//         C wrapper function to solve's Kepler's equation num times.  
// inputs: 
//         ph_ma:  pointer to beginning element of array of doubles containing mean anomaly in radians 
//         ph_ecc: pointer to beginning element of array of doubles containing eccentricity 
//         num:    integer size of input arrays 
//         ph_eccanom: pointer to beginning element of array of doubles eccentric anomaly in radians 
// outputs:
//         ph_eccanom: values overwritten with eccentric anomaly
// assumptions:
//         input mean anomalies between 0 and 2pi
//         input eccentricities between 0 and 1
//         all three arrays have at least num elements 
//
void keplereq_wrapper_c(double *ph_ma, double *ph_ecc, int num, double *ph_eccanom)
{
	int gpuid = init_cuda();
	// put vectors in thrust format from raw points
	thrust::host_vector<double> h_ecc(ph_ecc,ph_ecc+num);
	thrust::host_vector<double> h_ma(ph_ma,ph_ma+num);

	cutCreateTimer(&memoryTime);  	cutCreateTimer(&kernelTime);
	cutResetTimer(memoryTime);    	cutResetTimer(kernelTime);

	if(gpuid>=0)
	{
	cutStartTimer(memoryTime);
	// transfer input params to GPU
	thrust::device_vector<double> d_ecc = h_ecc;
	thrust::device_vector<double> d_ma = h_ma;
	// allocate mem on GPU
	thrust::device_vector<double> d_eccanom(num);
	hipDeviceSynchronize();
	cutStopTimer(memoryTime);
	
	// distribute the computation to the GPU
	cutStartTimer(kernelTime);
	thrust::for_each(
	   thrust::make_zip_iterator(thrust::make_tuple(d_ma.begin(),d_ecc.begin(),d_eccanom.begin())),
	   thrust::make_zip_iterator(thrust::make_tuple(d_ma.end(),  d_ecc.end(),  d_eccanom.end())), 
	   keplereq_functor() );
	hipDeviceSynchronize();
	cutStopTimer(kernelTime);

	// transfer results back to host
	cutStartTimer(memoryTime);
	thrust::copy(d_eccanom.begin(),d_eccanom.end(),ph_eccanom);
	hipDeviceSynchronize();
	cutStopTimer(memoryTime);
	}
	else
	{
	// distribute the computation to the CPU
	cutStartTimer(kernelTime);
	thrust::for_each(
	   thrust::make_zip_iterator(thrust::make_tuple(h_ma.begin(),h_ecc.begin(),ph_eccanom)),
	   thrust::make_zip_iterator(thrust::make_tuple(h_ma.end(),  h_ecc.end(),  ph_eccanom+num)), 
	   keplereq_functor() );
	cutStopTimer(kernelTime);	
	}
}


// demo program for how to use
// 	keplereq_wrapper_c(ph_ma,ph_ecc,num_eval,ph_eccanom);
// command line arguments:
//      number of eccentricities
//      number of mean anomalies
//      verbose (0, 1, 2)
// example:  ./keplereq.exe 4096 8192 0
// performance note:  
//      For just solving Kepler's equation, CPU<->GPU memory transfer overhead 
// 	is several times more expensive than the actual calculations.
//	So you might as well calculate it many times.
//      Eventually move more calculations onto GPU to amortize memory transfer
//      On GF100, 32M evals take a total of 256ms, of which 213ms is memory
//
int main(int argc, char **argv)
{
	// set size parameters from defaults or command line
	int num_ecc = 4096;
	int num_ma = 4096;
	int verbose = 0;
	{
	std::istringstream iss;
	if(argc>1)
		{
		iss.str(std::string (argv[1]));
		iss >> num_ecc;
		iss.clear();
		}
	if(argc>2)
		{
		iss.str(std::string (argv[2]));
		iss >> num_ma;
		iss.clear();
		}
	if(argc>3)
		{
		iss.str(std::string (argv[3]));
		iss >> verbose;
		iss.clear();
		}
	}
	int num_eval = num_ecc*num_ma;

	std::cerr << "# num_ecc = " << num_ecc << " num_meannom = " << num_ma << " verbose = " << verbose << "\n";

	// allocate host memory
	thrust::host_vector<double> h_ecc(num_eval);
	thrust::host_vector<double> h_ma(num_eval);
	thrust::host_vector<double> h_eccanom(num_eval);

	// initialize data on host 
	for(int i=0;i<num_ecc;++i)
		{
		for(int j=0;j<num_ma;++j)
			{	
			int k = i*num_ma+j;
			h_ecc[k] = 0.3;//static_cast<double>(i)/static_cast<double>(num_ecc);
			h_ma[k]  = 2.*M_PI*static_cast<double>(j)/static_cast<double>(num_ma);
			}
		}

	// optional check up on input values
	if(verbose>1)
	   {
		for(int i = 0; i < h_ecc.size(); i++)
        	   std::cout << "p[" << i << "] = " << h_ecc[i] << std::endl;

		for(int i = 0; i < h_ma.size(); i++)
		   std::cout << "z[" << i << "] = " << h_ma[i] << std::endl;
	   }

	// extract raw pointers to host memory to simulate what you'd get from IDL or another library
	double *ph_ecc = &h_ecc[0]; 
	double *ph_ma = &h_ma[0]; 
	double *ph_eccanom = &h_eccanom[0];

	// wrapper function that could be called from IDL
	keplereq_wrapper_c(ph_ma,ph_ecc,num_eval,ph_eccanom);

	// print results to verify that this worked (optional)
	if(verbose>0)
	   {
	   for(int i = 0; i < h_eccanom.size(); i++)
	      std::cout << i << ' ' << h_ecc[i] << ' ' << h_ma[i] << ' ' << ph_eccanom[i] << std::endl;	
	   }

	// report time spent on calculations and memory transfer
	std::cerr << "# Time for kernel: " << cutGetTimerValue(kernelTime) << " ms, Time for memory: " << cutGetTimerValue(memoryTime) << " ms, Total time: " << cutGetTimerValue(kernelTime)+cutGetTimerValue(memoryTime) << " ms \n"; 
	
}

extern "C" {
  #include <stdio.h>
  #include "idl_export.h"

  double keplereq_cuda(int argc, void *argv[]) {
    keplereq_wrapper_c((double *) argv[0], (double *) argv[1], (IDL_LONG64) argv[2], (double *) argv[3]);  
    return -1; 
  }
}
